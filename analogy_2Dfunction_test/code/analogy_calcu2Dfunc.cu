#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <cmath>

#include <fstream>
#include <iostream>
#include <vector>
#include <stdio.h>


#define M 8
#define N 10
#define PI 3.14159265358979323846
#define THREAD_NUM_IN_BLOCK 8

typedef hipDoubleComplex Complex;

extern __shared__ Complex sharedMemory[];

static void HandleError(hipError_t err,
	const char* file,
	int line) {
	if (err != hipSuccess) {
		printf("%s in  %s at line %d\n",
			hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

void outArrayToFile(const char* filename_r,const char* filename_i, hipDoubleComplex* array, int L, int K){
  std::ofstream outFile_real(filename_r);
  std::ofstream outFile_img(filename_i);
  if (outFile_real.is_open() && outFile_img.is_open())
  {
    for (int i = 0; i < 8; ++i)
    {
      for (int j = 0; j < (2 * L + 1) * (2 * K + 1); j++)
      {
        outFile_real << array[j].x << " ";
        outFile_img << array[j].y << " ";
      }
      outFile_real << std::endl;
      outFile_img << std::endl;
    }
    
    outFile_real.close();
    outFile_img.close();
    std::cout << "output to file finished...\n";
  }
  
}


__device__ void displayMatrix(int tx, int ty, int judgeA, int judgeB, hipDoubleComplex * matrix,int num_in_row, int roll, int col, const char* name){
  if (tx == judgeA && ty == judgeB)
  {
    printf("\n*********************** %s ***********************\n", name);
    for (int i = 0; i < roll; i++)
    {
      for (int j = 0; j < col; j++)
      {
        printf("(%.12f,%.12f) ",matrix[i * num_in_row + j].x, matrix[i * num_in_row + j].y);
      }
      printf("\n----------------------------------------------------------\n");
    }
    printf("\n************************************************************\n");
  }
}

/*
 * @brief  calculate RD array
 * 
 * @param dev_sig       input 8 channel signal
 * @param dev_window_h  input window function
 * @param dev_x_ref     input reference signal
 * @param dev_exp_val   input exp param
 * @param L             input r-d param
 * @param K             input r-d param
 * @param dev_rd_array  output 8 channel r-d result
 */
__global__ void calcu2DArray(hipDoubleComplex* dev_sig, hipDoubleComplex* dev_window_h,
	hipDoubleComplex* dev_x_ref, hipDoubleComplex* dev_exp_val, int L, int K, 
	hipDoubleComplex* dev_rd_array) {
    int global_tid_x = blockIdx.x * blockDim.x + threadIdx.x;
    int global_tid_y = blockIdx.y * blockDim.y + threadIdx.y;
    
    int minor_dim = (2 * L + 1) > (2 * K + 1) ? (2 * K + 1) : (2 * L + 1);
    int major_dim = (2 * L + 1) > (2 * K + 1) ? (2 * L + 1) : (2 * K + 1);
    if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0)
    {
      printf("major_dim: %d \n", major_dim);
      printf("minor_dim: %d \n", minor_dim);
    }

    Complex sum = make_hipDoubleComplex(0.0, 0.0);
    for (int i = 0; i < 8; i++)
    {
      for (int j = 0; j < minor_dim; j++)
      {
        for (int k = 0; k < N; k++)
        {
          Complex exp_elem = dev_exp_val[j * N + k];
          Complex window_elem = dev_window_h[k];
          Complex sig_elem = dev_sig[i * N + k];
          Complex x_ref_elem = dev_x_ref[global_tid_y * N + k];
          sum = hipCadd(sum, hipCmul(exp_elem, hipCmul(window_elem, hipCmul(sig_elem, x_ref_elem))));
        }
        int location = i * major_dim * minor_dim + j * major_dim + global_tid_y;
        printf("__global_tid_x __global_tid_y : %d, %d location: %d \n", global_tid_x, global_tid_y, location);
      }

    }
}


hipDoubleComplex* initVec(double* real, double* imag, int m, int n, bool flag) {
	hipDoubleComplex* array = new hipDoubleComplex[m * n];
  if (flag)
  {
    for (int i = 0; i < m * n; i++) {
      if (imag != NULL) {
        array[i] = make_hipDoubleComplex(real[i], imag[i]);
      } else {
        array[i] = make_hipDoubleComplex(real[i], 0);
      }
	  }
  } else {
    for (int i = 0; i < m * n; i++) {
      array[i].x = real[i];
      if (imag != NULL) {
        array[i].y = imag[i];
      } else {
        array[i].y = 0;
      }
	  }
  }
  return array;
}
  
hipDoubleComplex* initVec(double* real, unsigned int m, unsigned int n) {
	hipDoubleComplex* array = new hipDoubleComplex[m * n];
	for (int i = 0; i < m * n; i++) {
		array[i].x = real[i];
		array[i].y = 0;
	}
	return array;
}

void calcuTwoDimRd(double* sig, double* sig_imag,
	double* window_h,
	double* x_ref, double* x_ref_imag,
	unsigned int sig_m, unsigned int sig_n,
	unsigned int window_m, unsigned int window_n,
	unsigned int x_ref_m, unsigned int x_ref_n,
	int L, int K) {
	// make CPU varis
	hipDoubleComplex* sig_vec = initVec(sig, sig_imag, sig_m, sig_n, true);
	hipDoubleComplex* window_vec = initVec(window_h, window_m, window_n);
	hipDoubleComplex* x_ref_vec = initVec(x_ref, x_ref_imag, x_ref_m, x_ref_n, false);
  for (int i = 0; i < THREAD_NUM_IN_BLOCK; i++)
  {
    printf("x_ref_vec: %.8f, %.8f \n", x_ref_vec[i].x, x_ref_vec[i].y);
  }
  

	// init exp_val xref_val
	hipDoubleComplex* exp_val = new hipDoubleComplex[(2 * K + 1) * N];
	hipDoubleComplex* xref_val = new hipDoubleComplex[(2 * L + 1) * N];

	for (int i = -K; i <= K; i++) {
		for (int n = 0; n < N; n++) {
      double angle = static_cast<double>(2.0 * PI * i * (n + 1) / N);
			exp_val[n + (i + K) * N] = make_hipDoubleComplex(cos(angle), sin(angle));
		}
	}

	// define GPU varis
	hipDoubleComplex* dev_sig, * dev_x_ref, * dev_window_h, * dev_rd_array, * dev_exp_val;

	// malloc mems for GPU varis;
	HANDLE_ERROR(hipMalloc((void**)&dev_sig, 8 * N * sizeof(hipDoubleComplex)));
	HANDLE_ERROR(hipMalloc((void**)&dev_x_ref, (2 * L + 1) * N * sizeof(hipDoubleComplex)));
	HANDLE_ERROR(hipMalloc((void**)&dev_window_h, N * sizeof(hipDoubleComplex)));
  HANDLE_ERROR(hipMalloc((void**)&dev_exp_val, (2 * K + 1) * N * sizeof(hipDoubleComplex)));
	HANDLE_ERROR(hipMalloc((void**)&dev_rd_array, 8 * (2 * L + 1) * (2 * K + 1) * sizeof(hipDoubleComplex)));

	// copy data to GPU
	HANDLE_ERROR(hipMemcpy(dev_sig, sig_vec, 8 * N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_x_ref, x_ref_vec, (2 * L + 1) * N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_window_h, window_vec, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_exp_val, exp_val, (2 * K + 1) * N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

  int grid_y = ((2 * L + 1) > (2 * K + 1)) ? (2 * L + 1) : (2 * K + 1);
  
	// set kernel
  dim3 threads(THREAD_NUM_IN_BLOCK, THREAD_NUM_IN_BLOCK);
  dim3 grids((N + threads.x - 1) / threads.x, (grid_y + threads.y - 1) / threads.y);

	printf("threadNumInBlock: %d * %d \n", threads.x, threads.y);
	printf("grid demension %d * %d \n", grids.x, grids.y);
  printf("output array demension: %d * %d \n", 8, (2 * L + 1) * (2 * K + 1));
  
  // Allocate CUDA events that we'll use for timing
  hipEvent_t start;
  HANDLE_ERROR(hipEventCreate(&start));

  hipEvent_t stop;
  HANDLE_ERROR(hipEventCreate(&stop));

  // Record the start event
  HANDLE_ERROR(hipEventRecord(start, NULL));

  std::cout << "ready to enter the kernel" << std::endl;
  // kernel start 
	calcu2DArray <<< grids, threads, 
  THREAD_NUM_IN_BLOCK * (THREAD_NUM_IN_BLOCK * 3)* sizeof(Complex)
	>>> (dev_sig, dev_window_h, dev_x_ref, dev_exp_val, L, K, dev_rd_array);

	hipError_t cudaerr = hipDeviceSynchronize();
	if (cudaerr == hipSuccess)
		printf("kernel launch success!\n");
	else {
		printf("kernel launch failed with error \"%s\".\n",
		hipGetErrorString(cudaerr));
	}

  // Record the stop event
  HANDLE_ERROR(hipEventRecord(stop, NULL));

  // Wait for the stop event to complete
  HANDLE_ERROR(hipEventSynchronize(stop));

  float msecTotal = 0.0f;
  HANDLE_ERROR(hipEventElapsedTime(&msecTotal, start, stop));

  // Compute and print the performance
  float msecPerMatrixMul = msecTotal  / 1;
  // double flopsPerMatrixMul = 2.0 * static_cast<double>(dimsA.x) *
  //                            static_cast<double>(dimsA.y) *
  //                            static_cast<double>(dimsB.x);
  // double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) /
  //                    (msecPerMatrixMul / 1000.0f);

  printf(
    "Time= %.3f msec," \
    " WorkgroupSize= %u threads/block\n",
    msecPerMatrixMul,
    threads.x * threads.y);

  hipDoubleComplex* out = new hipDoubleComplex[8 * (2 * L + 1) * (2 * K + 1)];
	HANDLE_ERROR(hipMemcpy(out, dev_rd_array, 8 * (2 * L + 1) * (2 * K + 1) * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));	

  printf("output:\n");
  for (int i = 0; i < 1; ++i)
  {
    for (int j = 0; j < 5; ++j)
    {
      printf("(%.12f,%.12f) \n",out[i * N + j].x, out[i * N + j].y);
    }
    printf("\n");
  }

  outArrayToFile("rd_output_real.txt", "rd_output_img.txt", out, 4, 1);
  
	hipFree(dev_rd_array);
	hipFree(dev_sig);
	hipFree(dev_window_h);
	hipFree(dev_x_ref);
	hipFree(out);
	free(window_vec);
	free(x_ref_vec);
	free(sig_vec);
}

hipDoubleComplex* getVectFromFile(char* real_, char* imag_, int m, int n) {
	std::ifstream fileReal(real_);
	std::ifstream fileImag(imag_);
	hipDoubleComplex* out = new hipDoubleComplex[m * n];
	int i = 0;
	double real, imag;
	while (fileReal >> real || fileImag >> imag) {
		if (i >= m * n) {
			break;
		}
		out[i].x = real;
		out[i].y = imag;
		i++;
	}
	return out;
}

hipDoubleComplex* getVectFromFile(char* real_, int m, int n) {
	std::ifstream fileReal(real_);
	hipDoubleComplex* out = new hipDoubleComplex[m * n];
	int i = 0;
	double real;
	while (fileReal >> real) {
		if (i >= m * n) {
			break;
		}
		out[i].x = real;
		out[i].y = 0;
		i++;
	}
	std::cout << i << std::endl;
	return out;
}

double* getDoubleFromFile(const char* filename_, int m, int n) {
	std::ifstream fileIn(filename_);
	double* out = new double[m * n];
	double val;
	int i = 0;
	while (fileIn >> val) {
		if (i >= m * n) {
			break;
		}
		out[i] = val;
		i++;
	}
	printf("%s :", filename_);
	std::cout << i << std::endl;
	return out;
}


int main() {
	double* sig_real = getDoubleFromFile("sig_real.txt", 8, 10);
	double* sig_imag = getDoubleFromFile("sig_imag.txt", 8, 10);
	double* x_real = getDoubleFromFile("y_real.txt", 1, 10);
	double* x_imag = getDoubleFromFile("y_imag.txt", 1, 10);
	double* window_real = getDoubleFromFile("window_h.txt", 1, 10);
	calcuTwoDimRd(sig_real, sig_imag, window_real, x_real, x_imag, 8, 10, 1, 10, 1, 10, 4, 1);
	return 0;
}

